#include "hip/hip_runtime.h"
#include "naive_cuda_simulation.cuh"
#include "physics/gravitation.h"
#include "physics/mechanics.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "cuda_wrappers.cuh"

void NaiveCudaSimulation::allocate_device_memory(Universe& universe, void** d_weights, void** d_forces, void** d_velocities, void** d_positions){

}

void NaiveCudaSimulation::free_device_memory(void** d_weights, void** d_forces, void** d_velocities, void** d_positions){

}

void NaiveCudaSimulation::copy_data_to_device(Universe& universe, void* d_weights, void* d_forces, void* d_velocities, void* d_positions){

}

void NaiveCudaSimulation::copy_data_from_device(Universe& universe, void* d_weights, void* d_forces, void* d_velocities, void* d_positions){

}

__global__
void calculate_forces_kernel(std::uint32_t num_bodies, double2* d_positions, double* d_weights, double2* d_forces){

}

void NaiveCudaSimulation::calculate_forces(Universe& universe, void* d_positions, void* d_weights, void* d_forces){
    
}

__global__
void calculate_velocities_kernel(std::uint32_t num_bodies, double2* d_forces, double* d_weights, double2* d_velocities){

}

void NaiveCudaSimulation::calculate_velocities(Universe& universe, void* d_forces, void* d_weights, void* d_velocities){

}

__global__
void calculate_positions_kernel(std::uint32_t num_bodies, double2* d_velocities, double2* d_positions){

}

void NaiveCudaSimulation::calculate_positions(Universe& universe, void* d_velocities, void* d_positions){

}

void NaiveCudaSimulation::simulate_epochs(Plotter& plotter, Universe& universe, std::uint32_t num_epochs, bool create_intermediate_plots, std::uint32_t plot_intermediate_epochs){

}

__global__
void get_pixels_kernel(std::uint32_t num_bodies, double2* d_positions, std::uint8_t* d_pixels, std::uint32_t plot_width, std::uint32_t plot_height, double plot_bounding_box_x_min, double plot_bounding_box_x_max, double plot_bounding_box_y_min, double plot_bounding_box_y_max){

}

std::vector<std::uint8_t> NaiveCudaSimulation::get_pixels(std::uint32_t plot_width, std::uint32_t plot_height, BoundingBox plot_bounding_box, void* d_positions, std::uint32_t num_bodies){
    std::vector<std::uint8_t> pixels;
    return pixels;
}

__global__
void compress_pixels_kernel(std::uint32_t num_raw_pixels, std::uint8_t* d_raw_pixels, std::uint8_t* d_compressed_pixels){

}

void NaiveCudaSimulation::compress_pixels(std::vector<std::uint8_t>& raw_pixels, std::vector<std::uint8_t>& compressed_pixels){

}

void NaiveCudaSimulation::simulate_epoch(Plotter& plotter, Universe& universe, bool create_intermediate_plots, std::uint32_t plot_intermediate_epochs, void* d_weights, void* d_forces, void* d_velocities, void* d_positions){
    calculate_forces(universe, d_positions, d_weights, d_forces);
    calculate_velocities(universe, d_forces, d_weights, d_velocities);
    calculate_positions(universe, d_velocities, d_positions);

    universe.current_simulation_epoch++;
    if(create_intermediate_plots){
        if(universe.current_simulation_epoch % plot_intermediate_epochs == 0){
            std::vector<std::uint8_t> pixels = get_pixels(plotter.get_plot_width(), plotter.get_plot_height(), plotter.get_plot_bounding_box(), d_positions, universe.num_bodies);
            plotter.add_active_pixels_to_image(pixels);

            // This is a dummy to use compression in plotting, although not beneficial performance-wise
            // ----
            // std::vector<std::uint8_t> compressed_pixels;
            // compressed_pixels.resize(pixels.size()/8);
            // compress_pixels(pixels, compressed_pixels);
            // plotter.add_compressed_pixels_to_image(compressed_pixels);
            // ----

            plotter.write_and_clear();
        }
    }
}

void NaiveCudaSimulation::calculate_forces_kernel_test_adapter(std::uint32_t grid_dim, std::uint32_t block_dim, std::uint32_t num_bodies, void* d_positions, void* d_weights, void* d_forces){
    // adapter function used by automatic tests. DO NOT MODIFY.
    dim3 blockDim(block_dim);
    dim3 gridDim(grid_dim);
    calculate_forces_kernel<<<gridDim, blockDim>>>(num_bodies, (double2*) d_positions, (double*) d_weights, (double2*) d_forces);
}

void NaiveCudaSimulation::calculate_velocities_kernel_test_adapter(std::uint32_t grid_dim, std::uint32_t block_dim, std::uint32_t num_bodies, void* d_forces, void* d_weights, void* d_velocities){
    // adapter function used by automatic tests. DO NOT MODIFY.
    dim3 blockDim(block_dim);
    dim3 gridDim(grid_dim);
    calculate_velocities_kernel<<<gridDim, blockDim>>>(num_bodies, (double2*) d_forces, (double*) d_weights, (double2*) d_velocities);
}

void NaiveCudaSimulation::calculate_positions_kernel_test_adapter(std::uint32_t grid_dim, std::uint32_t block_dim, std::uint32_t num_bodies, void* d_velocities, void* d_positions){
    // adapter function used by automatic tests. DO NOT MODIFY.
    dim3 blockDim(block_dim);
    dim3 gridDim(grid_dim);
    calculate_positions_kernel<<<gridDim, blockDim>>>(num_bodies, (double2*) d_velocities, (double2*) d_positions);
}
