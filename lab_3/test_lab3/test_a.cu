#include "hip/hip_runtime.h"
#include "test.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "structures/universe.h"

#include "input_generator/input_generator.h"

#include "test_a.cuh"


void fill_and_check_device(Universe& universe, void* d_weights, void* d_forces, void* d_velocities, void* d_positions){
    double* weights = new double[universe.num_bodies];
    double2* forces = new double2[universe.num_bodies];
    double2* velocities = new double2[universe.num_bodies];
    double2* positions = new double2[universe.num_bodies];

    for(std::uint32_t i = 0; i < universe.num_bodies; i++){
        weights[i] = (double) i;
        forces[i].x = (double)i;
        forces[i].y = (double) 2*i;
        velocities[i].x = (double) i;
        velocities[i].y = (double) 3*i;
        positions[i].x = (double) i;
        positions[i].y = (double) 4*i;
    }

    double* weights_buffer = new double[universe.num_bodies];
    double2* forces_buffer = new double2[universe.num_bodies];
    double2* velocities_buffer = new double2[universe.num_bodies];
    double2* positions_buffer = new double2[universe.num_bodies];

    for(std::uint32_t i = 0; i < universe.num_bodies; i++){
        weights_buffer[i] = 0.0;
        forces_buffer[i].x = 0.0;
        forces_buffer[i].y = 0.0;
        velocities_buffer[i].x = 0.0;
        velocities_buffer[i].y = 0.0;
        positions_buffer[i].x = 0.0;
        positions_buffer[i].y = 0.0;
    }

    hipMemcpy(d_weights, weights, sizeof(double)*universe.num_bodies, hipMemcpyHostToDevice);
    hipMemcpy(d_forces, forces, sizeof(double2)*universe.num_bodies, hipMemcpyHostToDevice);
    hipMemcpy(d_velocities, velocities, sizeof(double2)*universe.num_bodies, hipMemcpyHostToDevice);
    hipMemcpy(d_positions, positions, sizeof(double2)*universe.num_bodies, hipMemcpyHostToDevice);

    hipMemcpy(weights_buffer, d_weights, sizeof(double)*universe.num_bodies, hipMemcpyDeviceToHost);
    hipMemcpy(forces_buffer, d_forces, sizeof(double2)*universe.num_bodies, hipMemcpyDeviceToHost);
    hipMemcpy(velocities_buffer, d_velocities, sizeof(double2)*universe.num_bodies, hipMemcpyDeviceToHost);
    hipMemcpy(positions_buffer, d_positions, sizeof(double2)*universe.num_bodies, hipMemcpyDeviceToHost);

    for(std::uint32_t i = 0; i < universe.num_bodies; i++){
        ASSERT_FLOAT_EQ(weights_buffer[i], weights[i]);
        ASSERT_FLOAT_EQ(forces_buffer[i].x, forces[i].x);
        ASSERT_FLOAT_EQ(forces_buffer[i].y, forces[i].y);
        ASSERT_FLOAT_EQ(velocities_buffer[i].x, velocities[i].x);
        ASSERT_FLOAT_EQ(velocities_buffer[i].y, velocities[i].y);
        ASSERT_FLOAT_EQ(positions_buffer[i].x, positions[i].x);
        ASSERT_FLOAT_EQ(positions_buffer[i].y, positions[i].y);
    }

    delete(weights);
    delete(forces);
    delete(velocities);
    delete(positions);

    delete(weights_buffer);
    delete(forces_buffer);
    delete(velocities_buffer);
    delete(positions_buffer);
}

