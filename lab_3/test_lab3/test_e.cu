#include "hip/hip_runtime.h"
#include "test.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "structures/universe.h"

#include "input_generator/input_generator.h"

#include "test_e.cuh"

#include "reference_implementations.cuh"

#include "simulation/naive_cuda_simulation.cuh"

void call_calculate_forces_kernel(Universe& universe, void* d_positions, void* d_weights, void* d_forces){
    // launch at least universe.num_bodies threads in total
    // calculate grid dimensions
    std::uint32_t block_dim = 512;
    std::uint32_t grid_dim;
    
    if(universe.num_bodies % block_dim == 0){
        grid_dim = universe.num_bodies / block_dim;
    }
    else{
        grid_dim = (universe.num_bodies - (universe.num_bodies % block_dim) + block_dim) / block_dim;
    }
     

    NaiveCudaSimulation::calculate_forces_kernel_test_adapter(grid_dim, block_dim, universe.num_bodies, (double2*) d_positions, (double*) d_weights, (double2*) d_forces);
    // wait for kernel to finish
    hipDeviceSynchronize();
}