#include "hip/hip_runtime.h"
#include "test.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "structures/universe.h"

#include "input_generator/input_generator.h"

#include "test_c.cuh"

void reference_copy_data_to_device(Universe& universe, void*  d_weights_reference, void* d_forces_reference, void* d_velocities_reference, void* d_positions_reference){
    for(std::uint32_t i = 0; i < universe.num_bodies; i++){
            hipMemcpy(&(((double*)d_weights_reference)[i]), &(universe.weights[i]), sizeof(double), hipMemcpyHostToDevice);    
            double2 dummy_force = make_double2(universe.forces[i][0], universe.forces[i][1]);
            hipMemcpy(&(((double2*)d_forces_reference)[i]), &dummy_force, sizeof(double2), hipMemcpyHostToDevice);
            double2 dummy_velocity = make_double2(universe.velocities[i][0], universe.velocities[i][1]);
            hipMemcpy(&(((double2*)d_velocities_reference)[i]), &dummy_velocity, sizeof(double2), hipMemcpyHostToDevice);
            double2 dummy_position = make_double2(universe.positions[i][0], universe.positions[i][1]);
            hipMemcpy(&(((double2*)d_positions_reference)[i]), &dummy_position, sizeof(double2), hipMemcpyHostToDevice);
        }
}