#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "test.cuh"


void get_device_count(std::int32_t* device_count){  
    hipGetDeviceCount(device_count);
}