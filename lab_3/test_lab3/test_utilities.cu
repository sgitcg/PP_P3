#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "test_utilities.cuh"

#include <cstdint>

bool compare_device_double(void* d_double_ptr1, void* d_double_ptr2){
    double buffer_1 = 0.0;
    double buffer_2 = 0.0;

    hipMemcpy(&buffer_1, d_double_ptr1, sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(&buffer_2, d_double_ptr2, sizeof(double), hipMemcpyDeviceToHost);    

    return buffer_1 == buffer_2;
}

bool compare_device_doublevector(void* d_ptr1, void* d_ptr2, size_t size){
    double* buffer_1 = (double*) malloc(sizeof(double)*size);
    double* buffer_2 = (double*) malloc(sizeof(double)*size);

    hipMemcpy(buffer_1, d_ptr1, sizeof(double)*size, hipMemcpyDeviceToHost);
    hipMemcpy(buffer_2, d_ptr2, sizeof(double)*size, hipMemcpyDeviceToHost);

    bool ret_val = true;

    for(std::int32_t i = 0; i < size; i++){
        if(buffer_1[i] != buffer_2[i]){
            ret_val = false;
            break;
        }
    }

    free(buffer_1);
    free(buffer_2);

    return ret_val;
}

bool compare_device_double2vector(void* d_ptr1, void* d_ptr2, size_t size){
    double2* buffer_1 = (double2*) malloc(sizeof(double2)*size);
    double2* buffer_2 = (double2*) malloc(sizeof(double2)*size);

    hipMemcpy(buffer_1, d_ptr1, sizeof(double2)*size, hipMemcpyDeviceToHost);
    hipMemcpy(buffer_2, d_ptr2, sizeof(double2)*size, hipMemcpyDeviceToHost);    

    bool ret_val = true;

    for(std::int32_t i = 0; i < size; i++){
        if((buffer_1[i].x != buffer_2[i].x) || (buffer_2[i].y != buffer_2[i].y)){
            ret_val = false;
            break;
        }
    }

    free(buffer_1);
    free(buffer_2);

    return ret_val;
}