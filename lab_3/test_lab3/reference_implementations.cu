#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "reference_implementations.cuh"

#include "simulation/naive_cuda_simulation.cuh"

void reference_inflate_compressed_pixels(std::vector<std::uint8_t>& compressed_pixels, std::vector<std::uint8_t>& inflated_pixels){
    std::uint8_t bitmask_bit_0 = 1;
    std::uint8_t bitmask_bit_1 = 2;
    std::uint8_t bitmask_bit_2 = 4;
    std::uint8_t bitmask_bit_3 = 8;
    std::uint8_t bitmask_bit_4 = 16;
    std::uint8_t bitmask_bit_5 = 32;
    std::uint8_t bitmask_bit_6 = 64;
    std::uint8_t bitmask_bit_7 = 128;

    // inflate pixels
    for(std::uint8_t compressed_pixel : compressed_pixels){
        // unpack
        std::vector<bool> px_values;
        
        px_values.push_back(compressed_pixel & bitmask_bit_0);
        px_values.push_back(compressed_pixel & bitmask_bit_1);
        px_values.push_back(compressed_pixel & bitmask_bit_2);
        px_values.push_back(compressed_pixel & bitmask_bit_3);
        px_values.push_back(compressed_pixel & bitmask_bit_4);
        px_values.push_back(compressed_pixel & bitmask_bit_5);
        px_values.push_back(compressed_pixel & bitmask_bit_6);
        px_values.push_back(compressed_pixel & bitmask_bit_7);
        
        // store
        for(bool px_val : px_values){
            if(px_val){
                inflated_pixels.push_back(255);
            }
            else{
                inflated_pixels.push_back(0);
            }    
        }
    }
}
